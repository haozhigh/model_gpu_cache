#include <stdio.h>
#include <hip/hip_runtime.h>

#define DATA_TYPE long long

__global__ void read_cache(DATA_TYPE* device_array, int array_size) {
	int i;
    DATA_TYPE* j = &device_array[0];

	for (i = 0; i < array_size; i++)
    	j=*(DATA_TYPE**)j;

    device_array[0] = (DATA_TYPE)j;
}

int main(int argc, char* argv[]) {
    hipError_t err = hipSuccess;
    DATA_TYPE* host_array = NULL;
    DATA_TYPE* device_array = NULL;
    size_t size;
    int i;

    if (argc < 3) {
        printf("Not enough parameters! Exitting...\n");
        return -1;
    }
    int ARRAY_SIZE = atoi(argv[1]);
    int STRIDE = atoi(argv[2]);

    size = sizeof(DATA_TYPE) * ARRAY_SIZE;
    host_array = (DATA_TYPE*)malloc(size);
    if (host_array == NULL) {
        printf("Failed to malloc!\n");
        return -1;
    }

    err = hipMalloc((void**)&device_array, size);
    if (err != hipSuccess) {
        printf("Failed to cudaMalloc!\n");
        free(host_array);
        return -1;
    }

    for (i = 0; i < ARRAY_SIZE; i++) {
        DATA_TYPE t = i + STRIDE;
        if (t >= ARRAY_SIZE) t %= STRIDE;
        host_array[i] = (DATA_TYPE)device_array + (DATA_TYPE)sizeof(DATA_TYPE) * t;
    }

    err = hipMemcpy(device_array, host_array, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("Failed to cudaMemcpy!\n");
        free(host_array);
        hipFree(device_array);
        return -1;
    }

    read_cache<<<1, 1>>>(device_array, ARRAY_SIZE);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Failed to invoke kernel!\n");
        free(host_array);
        hipFree(device_array);
        return -1;
    }

    free(host_array);
    hipFree(device_array);
    return 0;
}
